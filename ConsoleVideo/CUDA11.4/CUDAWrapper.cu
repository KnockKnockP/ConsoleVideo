#include "hip/hip_runtime.h"
﻿#include "CUDAWrapper.cuh"
#include "RGB.h"
#include <cstdint>

/*
    CUDAWrapper.cuh
    RGB.h
    cstdint
*/

void Launch_DEVICE_GenerateFrame(const int blockCount,
                                 const int threadsPerBlock,
                                 const int xSize,
                                 const char *grayscaleCharacters,
                                 const int arraySize,
                                 const RGB *colors,
                                 char *frame) {
    DEVICE_GenerateFrame<<<blockCount, threadsPerBlock>>>(xSize, grayscaleCharacters, arraySize, colors, frame);
    return;
}
                                 
__global__ void DEVICE_GenerateFrame(const int xSize,
                                     const char *grayscaleCharacters,
                                     const int arraySize,
                                     const RGB *colors,
                                     char *frame) {
    int x = threadIdx.x,
        y = blockIdx.x;
    
    RGB color = colors[((xSize * y) + x)];
    int16_t average = ((color.r + color.g + color.b) / 3);
    
    float index = ((float)(average) / 255);
    index *= arraySize;
    index = roundf(index);
    
    if (index < 0) {
        index = 0;
    } else if (index >= arraySize) {
        index = (arraySize - 1);
    }
    
    frame[((xSize * y) + x)] = grayscaleCharacters[(int)(index)];
    return;
}